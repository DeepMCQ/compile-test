#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include"logger.h"
#include"parser_dense.h"
#include"parser.h"
#include"data.h"
#include"kernelgraph.h"
#include"config.h"

std::unique_ptr<Data> data;
std::unique_ptr<GraphWrapper> graph; 
int topk = 0;
int display_topk = 1;

void build_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
    data->add(idx,point);
    graph->add_vertex(idx,point);
}

std::vector<std::vector<std::pair<int,value_t>>> batch_queries;
std::vector<std::vector<idx_t>> results(ACC_BATCH_SIZE);

void flush_queries(){
	results.resize(batch_queries.size());
	const int repeat = 1; // NOTICE: You can repeat multiple times to have an average search performance
	for(int i = 0;i < repeat;++i)
	    graph->search_top_k_batch(batch_queries,topk,results);
    for(int i = 0;i < batch_queries.size();++i){
        auto& result = results[i];
        for(int i = 0;i < result.size() && i < display_topk;++i)
            printf("%zu ",result[i]);
        printf("\n");
    }
    batch_queries.clear();
}

void query_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
    batch_queries.push_back(point);
	// Uncomment the following lines to have a finer granularity batch processing
    //if(batch_queries.size() == ACC_BATCH_SIZE){
    //    flush_queries();
    //}
	/////////////////////
}




void usage(char** argv){
    printf("Usage: %s <build/test> <build_data> <query_data> <search_top_k> <row> <dim> <return_top_k> <l2/ip/cos>\n",argv[0]);
}

int main(int argc,char** argv){
    if(argc != 9){
        usage(argv);
        return 1;
    }
	// You may need to increase this parameter for some new GPUs
    hipDeviceSetLimit(hipLimitMallocHeapSize,800*1024*1024);
	//////////////////////
	size_t row = atoll(argv[5]);
	int dim = atoi(argv[6]);
	display_topk = atoi(argv[7]);
	std::string dist_type = argv[8];
	data = std::unique_ptr<Data>(new Data(row,dim));
	if(dist_type == "l2"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<0>(data.get())); 
	}else if(dist_type == "ip"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<1>(data.get())); 
	}else if(dist_type == "cos"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<2>(data.get())); 
	}else{
		usage(argv);
		return 1;
	}
    std::string mode = std::string(argv[1]);
    topk = atoi(argv[4]);
    if(mode == "build"){
        //std::unique_ptr<ParserDense> build_parser(new ParserDense(argv[2],build_callback));
        std::unique_ptr<Parser> build_parser(new Parser(argv[2],build_callback));
        fprintf(stderr,"Writing the graph and data...");    
        data->dump();
        fprintf(stderr,"...");    
        graph->dump();
        fprintf(stderr,"done\n");    
    }else if(mode == "test"){
        fprintf(stderr,"Loading the graph and data...");    
        data->load();
        fprintf(stderr,"...");    
        graph->load();
        fprintf(stderr,"done\n");    
        //std::unique_ptr<ParserDense> query_parser(new ParserDense(argv[3],query_callback));
        std::unique_ptr<Parser> query_parser(new Parser(argv[3],query_callback));
		flush_queries();	
    }else{
        usage(argv);
        return 1;
    }
    return 0;
}
